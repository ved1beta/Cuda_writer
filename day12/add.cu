#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>

__global__ void add(float* A , float* B , float* C , int n  ){
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if(i < n){
        C[i] = B[i] + A[i];
    }
}

int main(){
    const int N = 10;
    float A[N], B[N], C[N];

    // Initialize input arrays
    for (int i = 0; i < N; i++) {
        A[i] = i * 1.0f;  // Some sample values
        B[i] = i * 2.0f;
    }

    float *d_a , *d_b , *d_c ;

    hipMalloc(&d_a , N * sizeof(float));
    hipMalloc(&d_b , N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));

    hipMemcpy(d_a , A , N* sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b , B , N* sizeof(float), hipMemcpyHostToDevice);

    int blocksize = 256 ; 
    int gridsize = (N+ blocksize -1)/blocksize; 

    add<<<gridsize , blocksize>>>(d_a , d_b , d_c , N);

    hipError_t kernelError = hipGetLastError();
    if (kernelError != hipSuccess) {
        std::cerr << "Kernel launch error: " << hipGetErrorString(kernelError) << std::endl;
        return -1;
    }

    // Copy result back to host
    hipMemcpy(C, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Check for any CUDA errors
    hipError_t syncError = hipGetLastError();
    if (syncError != hipSuccess) {
        std::cerr << "CUDA error after kernel execution: " << hipGetErrorString(syncError) << std::endl;
        return -1;
    }

    // Print results
    std::cout << "Vector Addition Results:" << std::endl;
    for (int i = 0; i < N; i++) {
        std::cout << A[i] << " + " << B[i] << " = " << C[i] << std::endl;
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
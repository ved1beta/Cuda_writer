#include <iostream>
#include <cstdlib>
#include <cmath>
#include <cassert>  // Added this include for assert()
#include <hip/hip_runtime.h>

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

// Error checking macro
#define CHECK_CUDA_ERROR(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error in " << __FILE__ << " line " << __LINE__ << ": " \
                  << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
} while(0)

// The kernel from your code
template <const int BM, const int BN, const int BK, const int TM, const int TN>
__global__ void __launch_bounds__((BM * BN) / (TM * TN), 1)
    sgemm2DBlocktiling(int M, int N, int K, float alpha, const float *A,
                       const float *B, float beta, float *C) {
  const uint cRow = blockIdx.y;
  const uint cCol = blockIdx.x;
  const uint totalResultsBlocktile = BM * BN;
  // A thread is responsible for calculating TM*TN elements in the blocktile
  const uint numThreadsBlocktile = totalResultsBlocktile / (TM * TN);
  // ResultsPerBlock / ResultsPerThread == ThreadsPerBlock
  assert(numThreadsBlocktile == blockDim.x);
  // BN/TN are the number of threads to span a column
  const int threadCol = threadIdx.x % (BN / TN);
  const int threadRow = threadIdx.x / (BN / TN);
  // allocate space for the current blocktile in smem
  __shared__ float As[BM * BK];
  __shared__ float Bs[BK * BN];
  // Move blocktile to beginning of A's row and B's column
  A += cRow * BM * K;
  B += cCol * BN;
  C += cRow * BM * N + cCol * BN;
  // calculating the indices that this thread will load into SMEM
  const uint innerRowA = threadIdx.x / BK;
  const uint innerColA = threadIdx.x % BK;
  // calculates the number of rows of As that are being loaded in a single step
  // by a single block
  const uint strideA = numThreadsBlocktile / BK;
  const uint innerRowB = threadIdx.x / BN;
  const uint innerColB = threadIdx.x % BN;
  // for both As and Bs we want each load to span the full column-width, for
  // better GMEM coalescing (as opposed to spanning full row-width and iterating
  // across columns)
  const uint strideB = numThreadsBlocktile / BN;
  // allocate thread-local cache for results in registerfile
  float threadResults[TM * TN] = {0.0};
  // register caches for As and Bs
  float regM[TM] = {0.0};
  float regN[TN] = {0.0};
  // outer-most loop over block tiles
  for (uint bkIdx = 0; bkIdx < K; bkIdx += BK) {
    // populate the SMEM caches
    for (uint loadOffset = 0; loadOffset < BM; loadOffset += strideA) {
      As[(innerRowA + loadOffset) * BK + innerColA] =
          A[(innerRowA + loadOffset) * K + innerColA];
    }
    for (uint loadOffset = 0; loadOffset < BK; loadOffset += strideB) {
      Bs[(innerRowB + loadOffset) * BN + innerColB] =
          B[(innerRowB + loadOffset) * N + innerColB];
    }
    __syncthreads();
    // advance blocktile
    A += BK;     // move BK columns to right
    B += BK * N; // move BK rows down
    // calculate per-thread results
    for (uint dotIdx = 0; dotIdx < BK; ++dotIdx) {
      // block into registers
      for (uint i = 0; i < TM; ++i) {
        regM[i] = As[(threadRow * TM + i) * BK + dotIdx];
      }
      for (uint i = 0; i < TN; ++i) {
        regN[i] = Bs[dotIdx * BN + threadCol * TN + i];
      }
      for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
        for (uint resIdxN = 0; resIdxN < TN; ++resIdxN) {
          threadResults[resIdxM * TN + resIdxN] +=
              regM[resIdxM] * regN[resIdxN];
        }
      }
    }
    __syncthreads();
  }
  // write out the results
  for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
    for (uint resIdxN = 0; resIdxN < TN; ++resIdxN) {
      C[(threadRow * TM + resIdxM) * N + threadCol * TN + resIdxN] =
          alpha * threadResults[resIdxM * TN + resIdxN] +
          beta * C[(threadRow * TM + resIdxM) * N + threadCol * TN + resIdxN];
    }
  }
}

// Wrapper function from your code
void runSgemm2DBlocktiling(int M, int N, int K, float alpha, float *A, float *B,
                           float beta, float *C) {
  const uint BK = 8;
  const uint TM = 8;
  const uint TN = 8;
  if (M >= 128 && N >= 128) {
    const uint BM = 128;
    const uint BN = 128;
    dim3 gridDim(CEIL_DIV(N, BN), CEIL_DIV(M, BM));
    dim3 blockDim((BM * BN) / (TM * TN));
    sgemm2DBlocktiling<BM, BN, BK, TM, TN>
        <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
  } else {
    // this is a hacky solution to the underlying problem
    // of not having proper bounds checking in the kernel
    const uint BM = 64;
    const uint BN = 64;
    dim3 gridDim(CEIL_DIV(N, BN), CEIL_DIV(M, BM));
    dim3 blockDim((BM * BN) / (TM * TN));
    sgemm2DBlocktiling<BM, BN, BK, TM, TN>
        <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
  }
}

// Sequential CPU matrix multiplication for verification
void cpuSgemm(int M, int N, int K, float alpha, const float *A, const float *B, 
              float beta, float *C) {
  for (int i = 0; i < M; ++i) {
    for (int j = 0; j < N; ++j) {
      float sum = 0.0f;
      for (int k = 0; k < K; ++k) {
        sum += A[i * K + k] * B[k * N + j];
      }
      C[i * N + j] = alpha * sum + beta * C[i * N + j];
    }
  }
}

// Verify results
bool verifyResults(const float *cpuC, const float *gpuC, int M, int N, float tolerance = 1e-5) {
  for (int i = 0; i < M; ++i) {
    for (int j = 0; j < N; ++j) {
      float diff = std::abs(cpuC[i * N + j] - gpuC[i * N + j]);
      if (diff > tolerance) {
        std::cout << "Mismatch at position (" << i << ", " << j << "): " 
                  << cpuC[i * N + j] << " vs " << gpuC[i * N + j] << std::endl;
        return false;
      }
    }
  }
  return true;
}

int main() {
  // Test with matrix dimensions that are multiples of block sizes
  const int M = 256;
  const int N = 256;
  const int K = 256;
  
  // Host memory
  float *h_A = new float[M * K];
  float *h_B = new float[K * N];
  float *h_C = new float[M * N];
  float *h_C_ref = new float[M * N];
  
  // Initialize matrices with random values
  for (int i = 0; i < M * K; ++i) {
    h_A[i] = static_cast<float>(rand()) / RAND_MAX;
  }
  for (int i = 0; i < K * N; ++i) {
    h_B[i] = static_cast<float>(rand()) / RAND_MAX;
  }
  for (int i = 0; i < M * N; ++i) {
    h_C[i] = 0.0f;
    h_C_ref[i] = 0.0f;
  }
  
  // Device memory
  float *d_A, *d_B, *d_C;
  CHECK_CUDA_ERROR(hipMalloc(&d_A, M * K * sizeof(float)));
  CHECK_CUDA_ERROR(hipMalloc(&d_B, K * N * sizeof(float)));
  CHECK_CUDA_ERROR(hipMalloc(&d_C, M * N * sizeof(float)));
  
  // Copy data to device
  CHECK_CUDA_ERROR(hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(d_C, h_C, M * N * sizeof(float), hipMemcpyHostToDevice));
  
  // Parameters for GEMM
  float alpha = 1.0f;
  float beta = 0.0f;
  
  // Create CUDA events for timing
  hipEvent_t start, stop;
  CHECK_CUDA_ERROR(hipEventCreate(&start));
  CHECK_CUDA_ERROR(hipEventCreate(&stop));
  
  // Start timing
  CHECK_CUDA_ERROR(hipEventRecord(start));
  
  // Run our optimized GEMM
  runSgemm2DBlocktiling(M, N, K, alpha, d_A, d_B, beta, d_C);
  
  // Stop timing
  CHECK_CUDA_ERROR(hipEventRecord(stop));
  CHECK_CUDA_ERROR(hipEventSynchronize(stop));
  
  // Calculate elapsed time
  float milliseconds = 0;
  CHECK_CUDA_ERROR(hipEventElapsedTime(&milliseconds, start, stop));
  
  // Copy results back to host
  CHECK_CUDA_ERROR(hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));
  
  // Compute reference solution on CPU for verification
  cpuSgemm(M, N, K, alpha, h_A, h_B, beta, h_C_ref);
  
  // Verify results
  bool correct = verifyResults(h_C_ref, h_C, M, N);
  
  // Report results
  if (correct) {
    std::cout << "Test PASSED!" << std::endl;
    
    // Calculate and report performance
    float gflops = (2.0f * M * N * K) / (milliseconds * 1e6);
    std::cout << "Matrix dimensions: " << M << "x" << N << "x" << K << std::endl;
    std::cout << "Performance: " << gflops << " GFLOPS" << std::endl;
    std::cout << "Execution time: " << milliseconds << " ms" << std::endl;
  } else {
    std::cout << "Test FAILED!" << std::endl;
  }
  
  // Clean up
  delete[] h_A;
  delete[] h_B;
  delete[] h_C;
  delete[] h_C_ref;
  CHECK_CUDA_ERROR(hipFree(d_A));
  CHECK_CUDA_ERROR(hipFree(d_B));
  CHECK_CUDA_ERROR(hipFree(d_C));
  CHECK_CUDA_ERROR(hipEventDestroy(start));
  CHECK_CUDA_ERROR(hipEventDestroy(stop));
  
  return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure we don't access beyond array bounds
    if (tid < n) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    // Vector size
    int n = 1000000;
    size_t size = n * sizeof(float);
    
    // Host vectors
    float *h_a, *h_b, *h_c;
    
    // Device vectors
    float *d_a, *d_b, *d_c;
    
    // Allocate memory for host vectors
    h_a = (float*)malloc(size);
    h_b = (float*)malloc(size);
    h_c = (float*)malloc(size);
    
    // Initialize host vectors
    for (int i = 0; i < n; i++) {
        h_a[i] = rand() / (float)RAND_MAX;
        h_b[i] = rand() / (float)RAND_MAX;
    }
    
    // Allocate memory for device vectors
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);
    
    // Copy host vectors to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    
    // Define thread block and grid dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    
    // Launch kernel
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);
    
    // Wait for GPU to finish
    hipDeviceSynchronize();
    
    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    
    // Verify result
    for (int i = 0; i < n; i++) {
        if (fabs(h_a[i] + h_b[i] - h_c[i]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            break;
        }
    }
    
    printf("Test PASSED\n");
    
    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    // Free host memory
    free(h_a);
    free(h_b);
    free(h_c);
    
    return 0;
}
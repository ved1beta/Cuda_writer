
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

__global__ void vecadd(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    const int N = 10; 
    float A[N], B[N], C[N];

    // Initialize input arrays
    for (int i = 0; i < N; i++) {
        A[i] = i * 1.0f;  // Some sample values
        B[i] = i * 2.0f;
    }

    // Device pointers
    float *d_a, *d_b, *d_c;

    // Allocate device memory
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_a, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, N * sizeof(float), hipMemcpyHostToDevice);

    // Configure kernel launch
    int blocksize = 256;
    int gridsize = (N + blocksize - 1) / blocksize;  // Correct grid size calculation

    // Launch kernel
    vecadd<<<gridsize, blocksize>>>(d_a, d_b, d_c, N);

    // Check for kernel launch errors
    hipError_t kernelError = hipGetLastError();
    if (kernelError != hipSuccess) {
        std::cerr << "Kernel launch error: " << hipGetErrorString(kernelError) << std::endl;
        return -1;
    }

    // Copy result back to host
    hipMemcpy(C, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Check for any CUDA errors
    hipError_t syncError = hipGetLastError();
    if (syncError != hipSuccess) {
        std::cerr << "CUDA error after kernel execution: " << hipGetErrorString(syncError) << std::endl;
        return -1;
    }

    // Print results
    std::cout << "Vector Addition Results:" << std::endl;
    for (int i = 0; i < N; i++) {
        std::cout << A[i] << " + " << B[i] << " = " << C[i] << std::endl;
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}